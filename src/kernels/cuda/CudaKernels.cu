#include "hip/hip_runtime.h"
/**
 * @copyright (c) 2022 King Abdullah University of Science and Technology (KAUST).
 *                     All rights reserved.
 */

#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hcorepp/kernels/cuda/CudaKernels.hpp>
#include <iostream>
#include <cuda/std/type_traits>
#include "blas/util.hh"
#include "blas/scal.hh"
#include <hip/hip_complex.h>
#include <hcorepp/common/TypeCheck.hpp>
#include <hcorepp/kernels/memory.hpp>

#define THREADS 32
#define THREADS_1D 1024
const int max_blocks = 65535;

namespace hcorepp {
    namespace cudakernels {

        template<typename T>
        __global__ void GenerateIdentityMatrix_kernel(int64_t aNumOfCols, T *apMatrix) {
            int64_t x = blockIdx.x * blockDim.x + threadIdx.x;

            if (x >= aNumOfCols) {
                return;
            }

            int64_t arr_index = x * aNumOfCols + x;
            apMatrix[arr_index] = 1;

        }

        template<typename T>
        __global__ void
        MultiplyByAlpha_kernel(T *apArray, int64_t aRows, int64_t aCols, int64_t aM, int64_t aRank, T aAlpha) {
            int64_t x = blockIdx.x * blockDim.x + threadIdx.x;

            if (x >= aRows * aCols) {
                return;
            }
            apArray[aM * aRank + x] *= aAlpha;
        }

        template<typename T>
        __global__ void
        ProcessVPointer_kernel_with_Ungqr(int64_t aN, int64_t aCRank, int64_t Vm, T aBeta, T *apCV,
                                          int64_t aLdcV, T *V, int64_t aArank, const T *apBdata) {
            int64_t x = blockIdx.x * blockDim.x + threadIdx.x;
            int64_t y = blockIdx.y * blockDim.y + threadIdx.y;

            int64_t index = y * Vm + x;
            int64_t apCV_index = x * aLdcV + y;

            if (x >= aN || y >= aCRank) {
                return;
            }

            if (::cuda::std::is_same<T, hipFloatComplex>::value) {
//                hipFloatComplex temp = hipCmulf(aBeta, apCV[apCV_index]);
//                V[index] = (float2) hipConjf((hipFloatComplex) (aBeta * apCV[apCV_index]));
            } else if (::cuda::std::is_same<T, hipDoubleComplex>::value) {

            } else {
                V[index] = aBeta * apCV[apCV_index];
            }
        }

        template<typename T>
        __global__ void
        ProcessVPointer_kernel_without_Ungqr(int64_t aN, int64_t aCRank, int64_t Vm, T aBeta, T *apCV,
                                             int64_t aLdcV, T *V, int64_t aArank, const T *apBdata) {
            int64_t x = blockIdx.x * blockDim.x + threadIdx.x;
            int64_t y = blockIdx.y * blockDim.y + threadIdx.y;

            int64_t index = y * Vm + x;
            int64_t apCV_index = x * aLdcV + y;

            if (x >= aN || y >= aCRank) {
                return;
            }

            V[index] = aBeta * apCV[apCV_index];
        }

        template<typename T>
        __global__ void
        ProcessVPointer_kernel_with_Ungqr_part2(int64_t aN, int64_t aCRank, int64_t Vm, T aBeta, T *apCV,
                                                int64_t aLdcV, T *V, int64_t aArank, const T *apBdata) {
            int64_t x = blockIdx.x * blockDim.x + threadIdx.x;
            int64_t y = blockIdx.y * blockDim.y + threadIdx.y;

            int64_t index = y * Vm + x;
            int64_t apB_index = x * aArank + y;

            if (x >= aN || y >= aArank) {
                return;
            }
            T *Vptr = &V[aN * aCRank];
            if (::cuda::std::is_same<T, hipFloatComplex>::value) {

            } else if (::cuda::std::is_same<T, hipDoubleComplex>::value) {

            } else {

                Vptr[index] = apBdata[apB_index];
            }
        }

        template<typename T>
        __global__ void
        ProcessVPointer_kernel_without_Ungqr_part2(int64_t aN, int64_t aCRank, int64_t Vm, T aBeta, T *apCV,
                                                   int64_t aLdcV, T *V, int64_t aArank, const T *apBdata) {
            int64_t x = blockIdx.x * blockDim.x + threadIdx.x;
            int64_t y = blockIdx.y * blockDim.y + threadIdx.y;

            int64_t index = y * Vm + x;
            int64_t apB_index = x * aArank + y;

            if (x >= aN || y >= aArank) {
                return;
            }

            T *Vptr = &V[aN * aCRank];
            Vptr[index] = apBdata[apB_index];
        }

        template<typename T>
        __global__ void
        CalculateUVptrConj_kernel_(int64_t aRank, int64_t aVm, T *UVptr) {

            int64_t x = blockIdx.x * blockDim.x + threadIdx.x;
            int64_t y = blockIdx.y * blockDim.y + threadIdx.y;

            int64_t index = y * aRank + x;

            if (x >= aRank || y >= aVm) {
                return;
            }

            if (::cuda::std::is_same<T, hipFloatComplex>::value) {

            } else if (::cuda::std::is_same<T, hipDoubleComplex>::value) {

            } else {
                UVptr[index] = UVptr[index];
            }
        }

        template<typename T>
        __global__ void
        CalculateVTnew_kernel_with_Ungqr(int64_t aRkNew, bool aUngqr, int64_t aMinVmVn, blas::real_type<T> *apSigma,
                                         T *apVTnew, int64_t aSizeS, int64_t aVm) {
            int64_t x = blockIdx.x * blockDim.x + threadIdx.x;
            int64_t y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= aRkNew || y >= aMinVmVn) {
                return;
            }

            int64_t index = y * aSizeS;

            T alpha = apSigma[x];
            T *vt = &apVTnew[x];
            vt[index] *= alpha;
        }

        template<typename T>
        __global__ void
        CalculateVTnew_kernel_without_Ungqr(int64_t aRkNew, bool aUngqr, int64_t aMinVmVn, blas::real_type<T> *apSigma,
                                            T *apVTnew, int64_t aSizeS, int64_t aVm) {
            int64_t x = blockIdx.x * blockDim.x + threadIdx.x;
            int64_t y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= aRkNew || y >= aVm) {
                return;
            }

            int64_t index = y * aSizeS;

            T alpha = apSigma[x];
            T *vt = &apVTnew[x];
            vt[index] *= alpha;

            if (::cuda::std::is_same<T, hipFloatComplex>::value) {

            } else if (::cuda::std::is_same<T, hipDoubleComplex>::value) {

            } else {
                apVTnew[index] = apVTnew[index];
            }
        }

        template<typename T>
        __global__ void
        CalculateUVptr_kernel(int64_t aRank, int64_t aVm, T *UVptr, const T *Vnew) {
            int64_t x = blockIdx.x * blockDim.x + threadIdx.x;
            int64_t y = blockIdx.y * blockDim.y + threadIdx.y;
            if (x >= aRank || y >= aVm) {
                return;
            }

            int64_t uv_index = y * aRank + x;
            int64_t vnew_index = x * aVm + y;

            if (::cuda::std::is_same<T, hipFloatComplex>::value) {

            } else if (::cuda::std::is_same<T, hipDoubleComplex>::value) {

            } else {
                UVptr[uv_index] = Vnew[vnew_index];
            }

        }

        template<typename T>
        __global__ void
        CalculateNewRank_kernel_withSVD(int64_t aNewRank, blas::real_type<T> *apSigma, int64_t sizeS,
                                        blas::real_type<T> accuracy) {
            int64_t x = blockIdx.x * blockDim.x + threadIdx.x;

            if (x >= sizeS) {
                return;
            }
            blas::real_type<T> Sigma_0 = apSigma[0];

            if (x > 0) {
                if (apSigma[x] < accuracy * Sigma_0) {
                    Sigma_0 = apSigma[x];
                    aNewRank = x;
                    /// expected to have break statement here.
                }
            }
        }

        template<typename T>
        __global__ void
        CalculateNewRank_kernel_withoutSVD(int64_t aNewRank, blas::real_type<T> *apSigma, int64_t sizeS,
                                           blas::real_type<T> accuracy) {
            int64_t x = blockIdx.x * blockDim.x + threadIdx.x;

            if (x >= sizeS) {
                return;
            }

            if (x > 0) {
                if (apSigma[x] < accuracy) {
                    aNewRank = x;
                    /// expected to have break statement here.
                }
            }
        }

        template<typename T>
        static __device__
        void zlaset_lower_device(int m, int n, T offdiag, T diag, T *A, int lda) {
            int ind = blockIdx.x * THREADS + threadIdx.x;
            int iby = blockIdx.y * THREADS;
            /* check if full block-column && (below diag) */
            bool full = (iby + THREADS <= n && (ind >= iby + THREADS));
            /* do only rows inside matrix, and blocks not above diag */
            if (ind < m && ind + THREADS > iby) {
                A += ind + iby * lda;
                if (full) {
                    // full block-column, off-diagonal block
#pragma unroll
                    for (int j = 0; j < THREADS; ++j) {
                        A[j * lda] = offdiag;
                    }
                } else {
                    // either partial block-column or diagonal block
                    for (int j = 0; j < THREADS && iby + j < n; ++j) {
                        if (iby + j == ind)
                            A[j * lda] = diag;
                        else if (ind > iby + j)
                            A[j * lda] = offdiag;
                    }
                }
            }
        }

        template<typename T>
        __global__ void zlaset_lower_kernel(int m, int n, T offdiag, T diag, T *dA, int ldda) {
            zlaset_lower_device(m, n, offdiag, diag, dA, ldda);
        }

        template<typename T>
        static __device__
        void zlaset_upper_device(int m, int n, T offdiag, T diag, T *A, int lda) {
            int ind = blockIdx.x * THREADS + threadIdx.x;
            int iby = blockIdx.y * THREADS;
            /* check if full block-column && (above diag) */
            bool full = (iby + THREADS <= n && (ind + THREADS <= iby));
            /* do only rows inside matrix, and blocks not below diag */
            if (ind < m && ind < iby + THREADS) {
                A += ind + iby * lda;
                if (full) {
                    // full block-column, off-diagonal block
#pragma unroll
                    for (int j = 0; j < THREADS; ++j) {
                        A[j * lda] = offdiag;
                    }
                } else {
                    // either partial block-column or diagonal block
                    for (int j = 0; j < THREADS && iby + j < n; ++j) {
                        if (iby + j == ind)
                            A[j * lda] = diag;
                        else if (ind < iby + j)
                            A[j * lda] = offdiag;
                    }
                }
            }
        }

        template<typename T>
        __global__ void zlaset_upper_kernel(int m, int n, T offdiag, T diag, T *dA, int ldda) {
            zlaset_upper_device(m, n, offdiag, diag, dA, ldda);
        }

        template<typename T>
        static __device__
        void zlaset_full_device(int m, int n, T offdiag, T diag, T *A, int lda) {
            int ind = blockIdx.x * THREADS + threadIdx.x;
            int iby = blockIdx.y * THREADS;
            /* check if full block-column && (below diag || above diag || offdiag == diag) */
            bool full = (iby + THREADS <= n &&
                         (ind >= iby + THREADS || ind + THREADS <= iby || (offdiag == diag)));
//                         MAGMA_Z_EQUAL(offdiag, diag)));
            /* do only rows inside matrix */
            if (ind < m) {
                A += ind + iby * lda;
                if (full) {
                    // full block-column, off-diagonal block or offdiag == diag
#pragma unroll
                    for (int j = 0; j < THREADS; ++j) {
                        A[j * lda] = offdiag;
                    }
                } else {
                    // either partial block-column or diagonal block
                    for (int j = 0; j < THREADS && iby + j < n; ++j) {
                        if (iby + j == ind)
                            A[j * lda] = diag;
                        else
                            A[j * lda] = offdiag;
                    }
                }
            }
        }

        template<typename T>
        __global__ void zlaset_full_kernel(int m, int n, T offdiag, T diag, T *dA, int ldda) {
            zlaset_full_device(m, n, offdiag, diag, dA, ldda);
        }

        template<typename T>
        static __device__
        void zlacpy_lower_device(int m, int n, const T *dA, int ldda, T *dB, int lddb) {
            int BLK_X = blockDim.x; // THREADS
            int BLK_Y = blockDim.y;
            int ind = blockIdx.x * BLK_X + threadIdx.x;
            int iby = blockIdx.y * BLK_Y;
            /* check if full block-column && (below diag) */
            bool full = (iby + BLK_Y <= n && (ind >= iby + BLK_Y));
            /* do only rows inside matrix, and blocks not above diag */
            if (ind < m && ind + BLK_X > iby) {
                dA += ind + iby * ldda;
                dB += ind + iby * lddb;
                if (full) {
                    // full block-column, off-diagonal block
#pragma unroll
                    for (int j = 0; j < BLK_Y; ++j) {
                        dB[j * lddb] = dA[j * ldda];
                    }
                } else {
                    // either partial block-column or diagonal block
                    for (int j = 0; j < BLK_Y && iby + j < n && ind >= iby + j; ++j) {
                        dB[j * lddb] = dA[j * ldda];
                    }
                }
            }
        }

        template<typename T>
        __global__ void zlacpy_lower_kernel(int m, int n, const T *dA, int ldda, T *dB, int lddb) {
            zlacpy_lower_device(m, n, dA, ldda, dB, lddb);
        }

        template<typename T>
        __global__ void zlacpy_full_kernel(int m, int n, const T *dA, int ldda, T *dB, int lddb) {
            int BLK_X = blockDim.x;
            int BLK_Y = blockDim.y;
            int ind = blockIdx.x * BLK_X + threadIdx.x;
            int iby = blockIdx.y * BLK_Y + threadIdx.y;
            /* do only rows inside matrix */
            if (ind < m) {
                if (iby < n) {
                    dB[ind + iby * lddb] = dA[ind + iby * ldda];
                }
            }
        }

        template<typename T>
        __global__ void zlacpy_upper_kernel(int m, int n, const T *dA, int ldda, T *dB, int lddb) {
            int BLK_X = blockDim.x;
            int BLK_Y = blockDim.y;
            int ind = blockIdx.x * BLK_X + threadIdx.x;
            int iby = blockIdx.y * BLK_Y + threadIdx.y;
            /* check if full block-column && (above diag) */
            /* do only rows inside matrix, and blocks not below diag */
            if (ind < m && ind <= iby && iby < n) {
                dB[ind + iby * lddb] = dA[ind + iby * ldda];
            }
        }

        template<typename T>
        void HCoreCudaKernels<T>::GenerateIdentityMatrix(int64_t aNumOfCols, T *apMatrix,
                                                         kernels::RunContext &aContext) {
            dim3 dimBlock(THREADS_1D, 1);
            dim3 dimGrid((aNumOfCols + dimBlock.x - 1) / dimBlock.x);

            GenerateIdentityMatrix_kernel<<<dimGrid, dimBlock, 0, aContext.GetStream()>>>(aNumOfCols, apMatrix);
        }

        template<typename T>
        void HCoreCudaKernels<T>::MultiplyByAlpha(T *apArray, int64_t aRows, int64_t aCols, int64_t aM, int64_t aRank,
                                                  T &aAlpha, kernels::RunContext &aContext) {
            dim3 dimBlock(THREADS_1D, 1);
            dim3 dimGrid(((aRows * aCols) + dimBlock.x - 1) / dimBlock.x);

            MultiplyByAlpha_kernel<<<dimGrid, dimBlock, 0, aContext.GetStream()>>>(apArray, aRows, aCols, aM, aRank,
                                                                                   aAlpha);
        }

        template<typename T>
        void HCoreCudaKernels<T>::Geqrf(int64_t aM, int64_t aN, T *apA, int64_t aLdA, T *apTau,
                                        kernels::RunContext &aContext) {
            /// https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSOLVER/Xgeqrf/cusolver_Xgeqrf_example.cu
            size_t d_lwork = 0;     /* size of workspace */
            size_t h_lwork = 0;     /* size of workspace */
            void *h_work = nullptr; /* host workspace */

            /* step 3: query working space of geqrf */
            hipsolverDnXgeqrf_bufferSize(aContext.GetCusolverDnHandle(), NULL, aM, aN, traits<T>::cuda_data_type, apA,
                                        aLdA,
                                        traits<T>::cuda_data_type, apTau, traits<T>::cuda_data_type, &d_lwork,
                                        &h_lwork);
            auto d_work = aContext.RequestWorkBuffer(sizeof(T) * d_lwork);

            /* step 4: QR factorization */
            hipsolverDnXgeqrf(aContext.GetCusolverDnHandle(), NULL, aM, aN, traits<T>::cuda_data_type, apA, aLdA,
                             traits<T>::cuda_data_type,
                             apTau, traits<T>::cuda_data_type, d_work, d_lwork, h_work, h_lwork,
                             aContext.GetInfoPointer());
        }

        template<typename T>
        void
        HCoreCudaKernels<T>::ProcessVpointer(int64_t aN, int64_t aCRank, bool aGetUngqr, int64_t Vm, T &aBeta, T *apCV,
                                             int64_t aLdcV, T *V,
                                             int64_t aArank, const T *apBdata, kernels::RunContext &aContext) {

            dim3 dimBlock(THREADS, THREADS);
            dim3 dimGrid((aN + dimBlock.x - 1) / dimBlock.x, (aCRank + dimBlock.y - 1) / dimBlock.y);

            if (aGetUngqr) {
                ProcessVPointer_kernel_with_Ungqr<T><<<dimGrid, dimBlock, 0, aContext.GetStream()>>>(aN,
                                                                                                     aCRank, Vm, aBeta,
                                                                                                     apCV, aLdcV,
                                                                                                     V, aArank,
                                                                                                     apBdata);
            } else {
                ProcessVPointer_kernel_without_Ungqr<T><<<dimGrid, dimBlock, 0, aContext.GetStream()>>>(aN,
                                                                                                        aCRank, Vm,
                                                                                                        aBeta, apCV,
                                                                                                        aLdcV, V,
                                                                                                        aArank,
                                                                                                        apBdata);
            }

            dim3 dimBlock_(THREADS, THREADS);
            dim3 dimGrid_((aN + dimBlock.x - 1) / dimBlock.x, (aArank + dimBlock.y - 1) / dimBlock.y);

            if (aGetUngqr) {
                ProcessVPointer_kernel_with_Ungqr_part2<T><<<dimGrid_, dimBlock_, 0, aContext.GetStream()>>>(aN, aCRank,
                                                                                                             Vm, aBeta,
                                                                                                             apCV,
                                                                                                             aLdcV,
                                                                                                             V, aArank,
                                                                                                             apBdata);
            } else {
                ProcessVPointer_kernel_without_Ungqr_part2<T><<<dimGrid_, dimBlock_, 0, aContext.GetStream()>>>(aN,
                                                                                                                aCRank,
                                                                                                                Vm,
                                                                                                                aBeta,
                                                                                                                apCV,
                                                                                                                aLdcV,
                                                                                                                V,
                                                                                                                aArank,
                                                                                                                apBdata);
            }

        }

        template<typename T>
        void
        HCoreCudaKernels<T>::CalculateUVptrConj(int64_t aRank, int64_t aVm, T *UVptr, kernels::RunContext &aContext) {
            dim3 dimBlock(THREADS, THREADS);
            dim3 dimGrid((aRank + dimBlock.x - 1) / dimBlock.x, (aVm + dimBlock.y - 1) / dimBlock.y);
            CalculateUVptrConj_kernel_<<<dimGrid, dimBlock, 0, aContext.GetStream()>>>(aRank, aVm, UVptr);
        }

        template<typename T>
        void
        HCoreCudaKernels<T>::CalculateVTnew(int64_t aRkNew, bool aUngqr, int64_t aMinVmVn, blas::real_type<T> *apSigma,
                                            T *apVTnew,
                                            int64_t aSizeS, int64_t aVm, kernels::RunContext &aContext) {
            dim3 dimBlock(THREADS, THREADS);

            if (aUngqr) {
                dim3 dimGrid((aRkNew + dimBlock.x - 1) / dimBlock.x, (aMinVmVn + dimBlock.y - 1) / dimBlock.y);
                CalculateVTnew_kernel_with_Ungqr<<<dimGrid, dimBlock, 0, aContext.GetStream()>>>(aRkNew, aUngqr,
                                                                                                 aMinVmVn, apSigma,
                                                                                                 apVTnew,
                                                                                                 aSizeS, aVm);
            } else {
                dim3 dimGrid((aRkNew + dimBlock.x - 1) / dimBlock.x, (aVm + dimBlock.y - 1) / dimBlock.y);
                CalculateVTnew_kernel_without_Ungqr<<<dimGrid, dimBlock, 0, aContext.GetStream()>>>(aRkNew, aUngqr,
                                                                                                    aMinVmVn, apSigma,
                                                                                                    apVTnew,
                                                                                                    aSizeS, aVm);
            }
        }

        template<typename T>
        void HCoreCudaKernels<T>::CalculateUVptr(int64_t aRank, int64_t aVm, T *UVptr, const T *Vnew,
                                                 kernels::RunContext &aContext) {
            dim3 dimBlock(THREADS, THREADS);

            dim3 dimGrid((aRank + dimBlock.x - 1) / dimBlock.x, (aVm + dimBlock.y - 1) / dimBlock.y);

            CalculateUVptr_kernel<<<dimGrid, dimBlock, 0, aContext.GetStream()>>>(aRank, aVm, UVptr, Vnew);
        }

        template<typename T>
        void HCoreCudaKernels<T>::CalculateNewRank(int64_t &aNewRank, bool aTruncatedSvd, blas::real_type<T> *apSigma,
                                                   int64_t sizeS, blas::real_type<T> accuracy,
                                                   kernels::RunContext &aContext) {
            auto host_sigma = new blas::real_type<T>[sizeS];
            hcorepp::memory::Memcpy<blas::real_type<T>>(host_sigma, apSigma, sizeS, aContext,
                                                        memory::MemoryTransfer::DEVICE_TO_HOST);
            aContext.Sync();
            //TODO do a proper reduction kernel and memcpy only rank
//            aNewRank = sizeS;
//
//            dim3 dimBlock(THREADS);
//
//            dim3 dimGrid((sizeS + dimBlock.x - 1) / dimBlock.x);
//
//            if (aTruncatedSvd) {
//                CalculateNewRank_kernel_withSVD<T><<<dimGrid, dimBlock>>>(aNewRank, apSigma, sizeS, accuracy);
//            } else {
//                CalculateNewRank_kernel_withoutSVD<T><<<dimGrid, dimBlock>>>(aNewRank, apSigma, sizeS, accuracy);
//            }
            aNewRank = sizeS;
            if (aTruncatedSvd) {
                blas::real_type<T> Sigma_0 = host_sigma[0];
                for (int64_t i = 1; i < sizeS; i++) {
                    if (host_sigma[i] < accuracy * Sigma_0) {
                        Sigma_0 = host_sigma[i];
                        aNewRank = i;
                        break;
                    }
                }
            } else {
                for (int64_t i = 1; i < sizeS; i++) {
                    if (host_sigma[i] < accuracy) {
                        aNewRank = i;
                        break;
                    }
                }
            }
            delete[] host_sigma;

        }

        template<typename T>
        void
        HCoreCudaKernels<T>::SVD(common::Job aJobu, common::Job aJobvt, int64_t aM, int64_t aN, T *apA, int64_t aLdA,
                                 T *apS, T *apU,
                                 int64_t aLdU, T *apVT, int64_t aLdVt, common::CompressionType aSVDOperationType,
                                 kernels::RunContext &aContext) {
            ///https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSOLVER/Xgesvd/cusolver_Xgesvd_example.cu
            size_t d_lwork = 0;     /* size of workspace */
            size_t h_lwork = 0;     /* size of workspace */
            void *h_work = nullptr; /* host workspace */
            /* step 3: query working space of geqrf */
            cusolverDnXgesvd_bufferSize(aContext.GetCusolverDnHandle(), NULL, (signed char) aJobu, (signed char) aJobvt,
                                        aM, aN,
                                        traits<T>::cuda_data_type, apA, aLdA, traits<T>::cuda_data_type, apS,
                                        traits<T>::cuda_data_type, apU, aLdU, traits<T>::cuda_data_type, apVT, aLdVt,
                                        traits<T>::cuda_data_type, &d_lwork, &h_lwork);
            T *d_work = (T *) aContext.RequestWorkBuffer(sizeof(T) * d_lwork);

            /* step 4: compute SVD */
            cusolverDnXgesvd(aContext.GetCusolverDnHandle(), NULL, (signed char) aJobu, (signed char) aJobvt, aM, aN,
                             traits<T>::cuda_data_type, apA, aLdA, traits<T>::cuda_data_type, apS,
                             traits<T>::cuda_data_type, apU, aLdU, traits<T>::cuda_data_type, apVT, aLdVt,
                             traits<T>::cuda_data_type, d_work, d_lwork,
                             h_work, h_lwork, aContext.GetInfoPointer());
        }

        template<typename T>
        void
        HCoreCudaKernels<T>::Unmqr(common::SideMode aSide, common::BlasOperation aTrans, int64_t aM, int64_t aN,
                                   int64_t aK,
                                   T const *apA, int64_t aLdA, T const *apTau, T *apC, int64_t aLdC,
                                   kernels::RunContext &aContext) {
            ///https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSOLVER/ormqr/cusolver_ormqr_example.cu
            size_t d_lwork = 0;     /* size of workspace */
            /* step 3: query working space of geqrf */

            hipsolverDnDormqr_bufferSize(aContext.GetCusolverDnHandle(), (hipblasSideMode_t) aSide,
                                        (hipblasOperation_t) aTrans, aM, aN, aM,
                                        (const double *) apA, aLdA, (const double *) apTau, (const double *) apC, aLdC,
                                        (int *) &d_lwork);
            T *d_work = (T *) aContext.RequestWorkBuffer(sizeof(T) * d_lwork);
            hipsolverDnDormqr(aContext.GetCusolverDnHandle(), (hipblasSideMode_t) aSide, (hipblasOperation_t) aTrans, aM,
                             aN, aM,
                             (const double *) apA, aLdA, (const double *) apTau, (double *) apC, aLdC,
                             (double *) d_work, d_lwork, aContext.GetInfoPointer());
        }

        template<typename T>
        void HCoreCudaKernels<T>::Laset(common::MatrixType aMatrixType, int64_t aM, int64_t aN, T aOffdiag, T aDiag,
                                        T *apA, int64_t aLdA, kernels::RunContext &aContext) {

#define dA(i_, j_) (dA + (i_) + (j_)*ldda)


            int info = 0;
            if (aMatrixType != common::MatrixType::Lower && aMatrixType != common::MatrixType::Upper &&
                aMatrixType != common::MatrixType::General)
                info = -1;
            else if (aM < 0)
                info = -2;
            else if (aN < 0)
                info = -3;
            else if (aLdA < std::max((int64_t) 1, aM))
                info = -7;

            if (info != 0) {
//                magma_xerbla(__func__, -(info));
                return;  //info;
            }

            if (aM == 0 || aN == 0) {
                return;
            }

            const int super_NB = max_blocks * THREADS;

            dim3 super_grid(ceil(aM / super_NB), ceil(aN / super_NB));

            dim3 threads(THREADS, 1);
            dim3 grid;

            int64_t mm, nn;
            if (aMatrixType == common::MatrixType::Lower) {
                for (unsigned int i = 0; i < super_grid.x; ++i) {
                    mm = (i == super_grid.x - 1 ? aM % super_NB : super_NB);
                    grid.x = ceil(mm / THREADS);
                    for (unsigned int j = 0; j < super_grid.y && j <= i; ++j) {  // from left to diagonal
                        nn = (j == super_grid.y - 1 ? aN % super_NB : super_NB);
                        grid.y = ceil(nn / THREADS);
                        if (i == j) {  // diagonal super block
                            zlaset_lower_kernel<<< grid, threads, 0, aContext.GetStream() >>>(mm, nn, aOffdiag, aDiag,
                                                                                              &apA[i * super_NB, j *
                                                                                                                 super_NB],
                                                                                              aLdA);
                        } else {           // off diagonal super block
                            zlaset_full_kernel<<< grid, threads, 0, aContext.GetStream()>>>
                                    (mm, nn, aOffdiag, aOffdiag, &apA[i * super_NB, j * super_NB], aLdA);
                        }
                    }
                }
            } else if (aMatrixType == common::MatrixType::Upper) {
                for (unsigned int i = 0; i < super_grid.x; ++i) {
                    mm = (i == super_grid.x - 1 ? aM % super_NB : super_NB);
                    grid.x = ceil(mm / THREADS);
                    for (unsigned int j = i; j < super_grid.y; ++j) {  // from diagonal to right
                        nn = (j == super_grid.y - 1 ? aN % super_NB : super_NB);
                        grid.y = ceil(nn / THREADS);
                        if (i == j) {  // diagonal super block
                            zlaset_upper_kernel<<< grid, threads, 0, aContext.GetStream() >>>(mm, nn, aOffdiag, aDiag,
                                                                                              &apA[i * super_NB, j *
                                                                                                                 super_NB],
                                                                                              aLdA);
                        } else {           // off diagonal super block
                            zlaset_full_kernel<<< grid, threads, 0, aContext.GetStream()>>>(mm, nn, aOffdiag, aOffdiag,
                                                                                            &apA[i * super_NB, j *
                                                                                                               super_NB],
                                                                                            aLdA);
                        }
                    }
                }
            } else {
                for (unsigned int i = 0; i < super_grid.x; ++i) {
                    mm = (i == super_grid.x - 1 ? aM % super_NB : super_NB);
                    grid.x = ceil(mm / THREADS);
                    for (unsigned int j = 0; j < super_grid.y; ++j) {  // full row
                        nn = (j == super_grid.y - 1 ? aN % super_NB : super_NB);
                        grid.y = ceil(nn / THREADS);
                        if (i == j) {  // diagonal super block
                            zlaset_full_kernel<<< grid, threads, 0, aContext.GetStream()>>>(mm, nn, aOffdiag, aDiag,
                                                                                            &apA[i * super_NB, j *
                                                                                                               super_NB],
                                                                                            aLdA);
                        } else {           // off diagonal super block
                            zlaset_full_kernel<<< grid, threads, 0, aContext.GetStream() >>>(mm, nn, aOffdiag, aOffdiag,
                                                                                             &apA[i * super_NB, j *
                                                                                                                super_NB],
                                                                                             aLdA);
                        }
                    }
                }
            }
        }

        template<typename T>
        void
        HCoreCudaKernels<T>::LaCpy(common::MatrixType aType, int64_t aM, int64_t aN, T *apA, int64_t aLdA, T *apB,
                                   int64_t aLdB, kernels::RunContext &aContext) {
#define dA(i_, j_) (dA + (i_) + (j_)*ldda)
#define dB(i_, j_) (dB + (i_) + (j_)*lddb)

            int info = 0;
            if (aType != common::MatrixType::Lower && aType != common::MatrixType::Upper &&
                aType != common::MatrixType::General)
                info = -1;
            else if (aM < 0)
                info = -2;
            else if (aN < 0)
                info = -3;
            else if (aLdA < std::max((int64_t) 1, aM))
                info = -5;
            else if (aLdB < std::max((int64_t) 1, aM))
                info = -7;

            if (info != 0) {
                return;
            }

            if (aM == 0 || aN == 0) {
                return;
            }

            const int64_t super_NB = max_blocks * THREADS;
            double divider = super_NB;
            dim3 super_grid(ceil(aM / divider), ceil(aN / divider));
            dim3 threads(THREADS, THREADS);
            dim3 grid;

            int64_t mm, nn;
            if (aType == common::MatrixType::Lower) {
                for (unsigned int i = 0; i < super_grid.x; ++i) {
                    mm = (i == super_grid.x - 1 ? aM % super_NB : super_NB);
                    grid.x = ceil(mm / THREADS);
                    for (unsigned int j = 0; j < super_grid.y && j <= i; ++j) {  // from left to diagonal
                        nn = (j == super_grid.y - 1 ? aN % super_NB : super_NB);
                        grid.y = ceil(nn / THREADS);
                        if (i == j) {  // diagonal super block
                            dim3 threads(THREADS, 1);
                            zlacpy_lower_kernel<<< grid, threads, 0, aContext.GetStream()>>>
                                    (mm, nn, &apA[i * super_NB, j * super_NB], aLdA, &apB[i * super_NB, j * super_NB],
                                     aLdB);
                        } else {           // off diagonal super block
                            zlacpy_full_kernel <<< grid, threads, 0, aContext.GetStream() >>>
                                    (mm, nn, &apA[i * super_NB, j * super_NB], aLdA, &apB[i * super_NB, j * super_NB],
                                     aLdB);
                        }
                    }
                }
            } else if (aType == common::MatrixType::Upper) {
                for (unsigned int i = 0; i < super_grid.x; ++i) {
                    mm = (i == super_grid.x - 1 ? aM % super_NB : super_NB);
                    grid.x = (mm + THREADS - 1) / THREADS;
                    for (unsigned int j = i; j < super_grid.y; ++j) {  // from diagonal to right
                        nn = (j == super_grid.y - 1 ? aN % super_NB : super_NB);
                        grid.y = (nn + THREADS - 1) / THREADS;
                        if (i == j) {  // diagonal super block
                            zlacpy_upper_kernel<<< grid, threads, 0, aContext.GetStream()>>>
                                    (mm, nn, &apA[i * super_NB, j * super_NB], aLdA, &apB[i * super_NB, j * super_NB],
                                     aLdB);
                        } else {           // off diagonal super block
                            zlacpy_full_kernel <<< grid, threads, 0, aContext.GetStream() >>>
                                    (mm, nn, &apA[i * super_NB, j * super_NB], aLdA, &apB[i * super_NB, j * super_NB],
                                     aLdB);
                        }
                    }
                }
            } else {
                if (aLdA == aLdB) {
                    hipMemcpyAsync(apB, apA, aM * aN * sizeof(T), hipMemcpyDeviceToDevice, aContext.GetStream());
                } else {
                    for (unsigned int i = 0; i < super_grid.x; ++i) {
                        mm = (i == super_grid.x - 1 ? aM % super_NB : super_NB);
                        grid.x = (mm + THREADS - 1) / THREADS;
                        for (unsigned int j = 0; j < super_grid.y; ++j) {  // full row
                            nn = (j == super_grid.y - 1 ? aN % super_NB : super_NB);
                            grid.y = (nn + THREADS - 1) / THREADS;
                            zlacpy_full_kernel <<< grid, threads, 0, aContext.GetStream() >>>
                                    (mm, nn, &apA[i * super_NB, j * super_NB], aLdA, &apB[i * super_NB, j * super_NB],
                                     aLdB);
                        }
                    }
                }
            }
        }

        template<typename T>
        void
        HCoreCudaKernels<T>::ungqr(int64_t aM, int64_t aN, int64_t aK, T *apA, int64_t aLdA, T *apTau,
                                   kernels::RunContext &aContext) {
            ///https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSOLVER/orgqr/cusolver_orgqr_example.cu
            int lwork_orgqr = 0;
            int lwork = 0;

            /* step 3: query working space of geqrf and orgqr */
            if constexpr(is_complex<T>()) {
                if constexpr(is_complex_float<T>()) {
                    cusolverDnCorgqr_bufferSize(aContext.GetCusolverDnHandle(), aM, aN, aK, apA, aLdA, apTau,
                                                (int *) (&lwork_orgqr));
                } else {
                    cusolverDnZorgqr_bufferSize(aContext.GetCusolverDnHandle(), aM, aN, aK, apA, aLdA, apTau,
                                                (int *) (&lwork_orgqr));
                }
            } else {
                if constexpr(is_double<T>()) {
                    hipsolverDnDorgqr_bufferSize(aContext.GetCusolverDnHandle(), aM, aN, aK, apA, aLdA, apTau,
                                                (int *) (&lwork_orgqr));
                } else {
                    hipsolverDnSorgqr_bufferSize(aContext.GetCusolverDnHandle(), aM, aN, aK, apA, aLdA, apTau,
                                                (int *) (&lwork_orgqr));
                }
            }


            lwork = lwork_orgqr;

            T *d_work = (T *)aContext.RequestWorkBuffer(sizeof(T) * lwork);
            int *d_info = aContext.GetInfoPointer();
            /* step 5: compute Q */
            if constexpr(is_complex<T>()) {
                if constexpr(is_complex_float<T>()) {
                    cusolverDnCorgqr(aContext.GetCusolverDnHandle(), aM, aN, aK, apA, aLdA, apTau, d_work, lwork,
                                     d_info);
                } else {
                    cusolverDnZorgqr(aContext.GetCusolverDnHandle(), aM, aN, aK, apA, aLdA, apTau, d_work, lwork,
                                     d_info);
                }
            } else {
                if constexpr(is_double<T>()) {
                    hipsolverDnDorgqr(aContext.GetCusolverDnHandle(), aM, aN, aK, apA, aLdA, apTau, d_work, lwork,
                                     d_info);
                } else {
                    hipsolverDnSorgqr(aContext.GetCusolverDnHandle(), aM, aN, aK, apA, aLdA, apTau, d_work, lwork,
                                     d_info);
                }
            }
        }

        HCOREPP_INSTANTIATE_CLASS(HCoreCudaKernels)

    }
}